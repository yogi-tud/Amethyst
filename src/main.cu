#include "cuda_try.cuh"
#include "cuda_time.cuh"
#include <cstdint>
#include <cstddef>
#include "utils.cuh"
#include "fast_prng.cuh"
#include <cstring>

#include "common_code.cuh"
#include <bit_iterator.cuh>
bool do_validation = true;

int run_count = 2;
int device_id =0;
int bench_part =2;

int wantbits = 64;

size_t grid_size_min = 32;
size_t grid_size_max = 8192;
size_t block_size_min = 32;
size_t block_size_max = 1024;

#define DECLARE_AND_CALL(name)                                                                                                                       \
    void name(                                                                                                                                       \
        size_t element_count, uint64_t* h_data1, uint64_t* h_data2, uint64_t* h_output, uint64_t* d_data1, uint64_t* d_data2, uint64_t* d_output,    \
        size_t bitmask_size);                                                                                                                        \
    name(element_count, h_data1, h_data2, h_output, d_data1, d_data2, d_output, bitmask_size);

int main(int argc, char** argv)
{
    bit_iterator foo{NULL};
    foo++;
    size_t element_count = 1 << 27; //27 - 1 gib per array
    bool use_rng = true;

    int w_argc = argc - 1; // remaining arg count
    while (w_argc > 0) {
        char* w_arg = argv[argc - (w_argc--)]; // working arg
        char* n_arg = (w_argc > 0) ? argv[argc - w_argc] : NULL; // next arg
        if (strcmp(w_arg, "--nv") == 0) {
            do_validation = false;
        }
        else if (strcmp(w_arg, "-n") == 0) {
            w_argc--;
            element_count = atoi(n_arg);
            if (element_count == 0) {
                element_count = 1024;
            }
        }
        else if (strcmp(w_arg, "-p") == 0) {
            w_argc--;
            bench_part = atoi(n_arg);
        }
        else if (strcmp(w_arg, "-N") == 0) {
            w_argc--;
            element_count = 1 << atoi(n_arg);
            if (element_count == 1) {
                element_count = 1 << 10;
            }
        }
        else if (strcmp(w_arg, "-r") == 0) {
            w_argc--;
            run_count = atoi(n_arg);
            if (run_count == 0) {
                run_count = 4;
            }
        }
        else if (strcmp(w_arg, "-b") == 0) {
            w_argc--;
            wantbits = atoi(n_arg);
            if (wantbits == 0) {
                exit(0);
            }
        }
        else if (strcmp(w_arg, "-d") == 0) {
            w_argc--;
            device_id = atoi(n_arg);
            
        }
        else if (strcmp(w_arg, "--gsm") == 0) {
            w_argc--;
            grid_size_min = atoi(n_arg);
            if (grid_size_min == 0) {
                exit(0);
            }
        }
        else if (strcmp(w_arg, "--gsM") == 0) {
            w_argc--;
            grid_size_max = atoi(n_arg);
            if (grid_size_max == 0) {
                exit(0);
            }
        }
        else if (strcmp(w_arg, "--bsm") == 0) {
            w_argc--;
            block_size_min = atoi(n_arg);
            if (block_size_min == 0) {
                exit(0);
            }
        }
        else if (strcmp(w_arg, "--bsM") == 0) {
            w_argc--;
            block_size_max = atoi(n_arg);
            if (block_size_max == 0) {
                exit(0);
            }
        }
        else if (strcmp(w_arg, "--ff") == 0) {
            use_rng = false;
        }
        else if (!strcmp(w_arg, "--help") || !strcmp(w_arg, "-h")) {
            printf(
                "%s [OPTIONS]\n"
                "    -h | --help: print this help\n"
                "    --nv:        disable validation (on by default)\n"
                "    -n <int>:    number of elements (default %lu)\n"
                "    -N <int>:    like -n, but as power of two, given x will lead to 2^x elements\n"
                "    -r <int>:    number of repeat runs for each benchmark configuration (default %u)\n"
                "    -b <int>:    maximum number of bits to test for each element, minimum is always 1 (default %u)\n"
                "    -p <int>:    Bench is split into 2 parts. 0 is first, 1 is second, 2 is all "
                "    -gsm <int>:  minimum grid size to test\n"
                "    -gsM <int>:  maximum grid size to test\n"
                "    -bsm <int>:  minimum block size to test\n"
                "    -bsM <int>:  maximum block size to test\n"
                "    --ff <int>:  instead of using random input data, fill all input bytes with 0xff\n",
                argv[0], element_count, run_count, wantbits
            );
            exit(0);
        }
        else {
            printf("ignoring unknown argument: \"%s\"\n", w_arg);
        }
    }

    size_t data_size = element_count * sizeof(uint64_t);
    size_t bitmask_size = ((element_count + 64 - 1) / 64) * sizeof(uint64_t);
    size_t output_data_size = data_size * 4; // group by need 2 outputs and reduce by key need 2 more
    CUDA_TRY(hipSetDevice(device_id));  // pick GPU device
    // generate data (compressed)
    uint64_t* h_data1 = (uint64_t*)malloc(data_size);
    uint64_t* h_data2 = (uint64_t*)malloc(data_size);
    uint64_t* h_output = (uint64_t*)malloc(output_data_size);
    fast_prng rng(42);
    if (use_rng) {
        for (size_t i = 0; i < data_size / sizeof(uint64_t); i++) {
            h_data1[i] = ((uint64_t)rng.rand()) << 32 | (uint64_t)rng.rand();
            h_data2[i] = ((uint64_t)rng.rand()) << 32 | (uint64_t)rng.rand();
        }
    }
    else {
        memset(h_data1, 0xFF, data_size);
        memset(h_data2, 0xFF, data_size);
    }
   
   //TODO REMOVE, SET MASK TO ZERO FOR COMP STORE. ADD PARAMETER FOR COMP MASK BEHAVIOR
   memset(h_data2, 0x00, data_size);

    // memcpy to gpu
    uint64_t* d_data1;
    uint64_t* d_data2;
    uint64_t* d_output;
    
    CUDA_TRY(hipMalloc(&d_data1, data_size));
    CUDA_TRY(hipMalloc(&d_data2, data_size));
    CUDA_TRY(hipMalloc(&d_output, output_data_size));
    CUDA_TRY(hipMemcpy(d_data1, h_data1, data_size, hipMemcpyHostToDevice));
    CUDA_TRY(hipMemcpy(d_data2, h_data2, data_size, hipMemcpyHostToDevice));

    printf("approach;element_count;bits;block_count;thread_count;time_ms;throughput;run\n");
    fflush(stdout);

    //write a loop that iterates over d_data1 and summarizes it
     
    if(bench_part == 0 || bench_part == 2)
    {

    
    DECLARE_AND_CALL(bench_compressstore_baseline_cub);
   DECLARE_AND_CALL(bench_compressstore_slabs);

  //    DECLARE_AND_CALL(bench_groupby_sm_array);
   //  DECLARE_AND_CALL(bench_groupby_sort_cub); // FIXME

  //  DECLARE_AND_CALL(bench_hop_baseline);
  // DECLARE_AND_CALL(bench_hop_nogaps);
  //  DECLARE_AND_CALL(bench_hop_slabs);
   // DECLARE_AND_CALL(bench_binary_op_baseline);
    }

    if(bench_part == 1 || bench_part == 2)
    {
    
 //   DECLARE_AND_CALL(bench_binary_op_nogaps);
  //  DECLARE_AND_CALL(bench_binary_op_slabs);

  //  DECLARE_AND_CALL(bench_filter_baseline);
  //  DECLARE_AND_CALL(bench_filter_nogaps);
 //   DECLARE_AND_CALL(bench_filter_slabs);
    }



    free(h_data1);
    free(h_data2);
    free(h_output);
    hipFree(d_output);
    hipFree(d_data1);
    hipFree(d_data2);
}
