#include "config.cuh"
#include <kernels/filter_baseline.cuh>

template <size_t block_size, int bits>
float bench_filter_baseline(
    size_t element_count,
    uint64_t* h_data1,
    uint64_t* h_data2,
    uint64_t* h_output,
    uint64_t* d_data1,
    uint64_t* d_data2,
    uint64_t* d_output,
    size_t bitmask_size,
    size_t block_count,
    bool validate)
{
    // generate bitmask based on predacte(on gpu)
    float time;
    CUDA_QUICKTIME(&time, {
        kernel_filter_baseline<block_size, bits, FILTER_PREDICATE_OP<bits>><<<block_count, block_size>>>(d_data1, element_count, d_output);
    });
    CUDA_TRY(hipDeviceSynchronize());

    if (!validate) {
        return time; // turns off validation
    }

    // memcpy to cpu
    CUDA_TRY(hipMemcpy(h_output, d_output, bitmask_size, hipMemcpyDeviceToHost));

    // validate on cpu
    for (size_t i = 0; i < element_count; i++) {
        bool expected = FILTER_PREDICATE_OP<bits>(h_data1[i] & (((uint64_t)~0) >> (64 - bits)));
        bool result = get_bitmask_element<bits>(h_output, i);
        if (expected != result) {
            bit_print(h_data1[i] & (((uint64_t)~0) >> (64 - bits)), true, true, true, "\n");
            bit_print(h_output[i / 64], true, true, true, "\n");
            printf("validation fail on element %lu (expected %u, got %u)\n", i, expected, result);
            assert(false);
            exit(-1);
        }
    }
    return time;
}

struct filter_baseline_launcher {
    static constexpr bool uses_block_size = true;
    static constexpr size_t smem_usage(int bits, size_t block_size)
    {
        return sizeof(uint32_t) * block_size;
    }
    static float used_data_size(int bits, size_t element_count)
    {
        return element_count * sizeof(uint64_t);
    }
    template <typename bits, size_t block_size>
    static float call(
        size_t element_count,
        uint64_t* h_data1,
        uint64_t* h_data2,
        uint64_t* h_output,
        uint64_t* d_data1,
        uint64_t* d_data2,
        uint64_t* d_output,
        size_t bitmask_size,
        size_t block_count,
        bool validate)
    {
        return bench_filter_baseline<block_size, bits::value>(
            element_count, h_data1, h_data2, h_output, d_data1, d_data2, d_output, bitmask_size, block_count, validate);
    }
};

void bench_filter_baseline(
    size_t element_count,
    uint64_t* h_data1,
    uint64_t* h_data2,
    uint64_t* h_output,
    uint64_t* d_data1,
    uint64_t* d_data2,
    uint64_t* d_output,
    size_t bitmask_size)
{
    bench_elementstuffing_wrapper<filter_baseline_launcher, 64>::call(
        "filter_baseline", element_count, h_data1, h_data2, h_output, d_data1, d_data2, d_output, bitmask_size);
}
